#include "defines.cuh"
#include "function_defines.cuh"

#include "stdlib.h"
#include "stdio.h"

void UpdateWorldToDevice(World *h_w, World **d_w){

	/*  The data copy to device should be implemented from ground up. 
	/*	a pointer which is contained by a pointer should be allocated first
	*/

	World *temp = (World*)malloc(sizeof(World));

	hipMalloc(&(temp->vp),sizeof(ViewPlane));
	cudaCheckErrors("viewplane allocate failed");
	hipMemcpy((temp->vp),h_w->vp,sizeof(ViewPlane),hipMemcpyHostToDevice);
	cudaCheckErrors("viewplane copy failed");

	temp->backgroundColor = h_w->backgroundColor;

	/*  GeometricObject                   
	/*  struct GeometricObject has 2 types, sphere and plane, contains a pointer to material  
	/*  if any change, this comment should be updated;
	*/
	temp->numObject = h_w->numObject;

	temp->objects = (GeometricObject**)malloc( temp->numObject * sizeof(GeometricObject*));
	for(int i = 0 ; i < temp->numObject ; ++i ){
		int sizeOfObject;
		switch( h_w->objects[i]->type ){
		case GMO_TYPE_SPHERE:
			sizeOfObject = sizeof(Sphere);
			break;
		case GMO_TYPE_PLANE:
			sizeOfObject = sizeof(Plane);
			break;
		default:
			sizeOfObject = sizeof(GeometricObject);
			break;
		}
		GeometricObject *h_gmo = (GeometricObject*)malloc(sizeOfObject);
		memcpy(h_gmo,h_w->objects[i],sizeOfObject);

		int sizeOfMaterial;
		switch( h_w->objects[i]->material->type ){
		case MATERIAL_TYPE_MATTE:
			sizeOfMaterial = sizeof(Matte);
			break;
		case MATERIAL_TYPE_PHONG:
			sizeOfMaterial = sizeof(Phong);
			break;
		default:
			sizeOfMaterial = sizeof(Material);
			break;
		}
		Material *d_m;
		hipMalloc(&d_m,sizeOfMaterial);
		cudaCheckErrors("material allocate failed");
		hipMemcpy(d_m,h_w->objects[i]->material,sizeOfMaterial,hipMemcpyHostToDevice);
		cudaCheckErrors("material copy failed");

		h_gmo->material = d_m;

		hipMalloc(temp->objects + i,sizeOfObject);
		cudaCheckErrors("geometricobjects allocate failed");
		hipMemcpy(temp->objects[i],h_gmo,sizeOfObject,hipMemcpyHostToDevice);
		cudaCheckErrors("geometricobjects copy failed");

		free(h_gmo);
	}
	GeometricObject **d_obj;
	hipMalloc(&d_obj,temp->numObject * sizeof(GeometricObject*));
	cudaCheckErrors("geometricobject* allocate failed");
	hipMemcpy(d_obj,temp->objects,temp->numObject * sizeof(GeometricObject*),hipMemcpyHostToDevice);
	cudaCheckErrors("geometricobject* copy failed");

	free(temp->objects);
	temp->objects = d_obj;

	/* Camara 
	*/
	int sizeOfCamara;
	switch( h_w->camara->type ){
	case CAMARA_TYPE_PINHOLE:
		sizeOfCamara = sizeof(Pinhole);
		break;
	default:
		sizeOfCamara = sizeof(Camara);
		break;
	}
	hipMalloc(&(temp->camara),sizeOfCamara);
	cudaCheckErrors("camara allocate failed");
	hipMemcpy(temp->camara, h_w->camara,sizeOfCamara,hipMemcpyHostToDevice);
	cudaCheckErrors("camara copy allocate failed");

	/* Light 
	*/
	hipMalloc(&(temp->ambient),sizeof(Ambient));
	cudaCheckErrors("ambient allocate failed");
	hipMemcpy(temp->ambient,h_w->ambient, sizeof(Ambient),hipMemcpyHostToDevice);
	cudaCheckErrors("ambient copy failed");

	temp->numLight = h_w->numLight;

	temp->lights = (Light**)malloc(temp->numLight * sizeof(Light*));
	for( int i = 0 ; i < temp->numLight ; ++i ){
		int sizeOfLight;
		switch( h_w->lights[i]->type ){
		case LIGHT_TYPE_AMBIENT:
			sizeOfLight = sizeof(Ambient);
			break;
		case LIGHT_TYPE_POINTLIGHT:
			sizeOfLight = sizeof(PointLight);
			break;
		case LIGHT_TYPE_DIRECTIONAL:
			sizeOfLight = sizeof(Directional);
			break;
		default:
			sizeOfLight = sizeof(Light);
			break;
		}
		hipMalloc( temp->lights + i , sizeOfLight );
		cudaCheckErrors(" lights allocate failed ");
		hipMemcpy( temp->lights[i] , h_w->lights[i] , sizeOfLight , hipMemcpyHostToDevice);
		cudaCheckErrors(" lights copy failed ");
	}
	Light **lts;
	hipMalloc( &lts ,temp->numLight * sizeof(Light*) );
	cudaCheckErrors("light* allocate failed");
	hipMemcpy(lts , temp->lights , temp->numLight * sizeof(Light*),hipMemcpyHostToDevice);

	free(temp->lights);
	temp->lights = lts;

	hipMalloc(d_w,sizeof(World));
	cudaCheckErrors("world allocate failed");
	hipMemcpy(*d_w,temp,sizeof(World),hipMemcpyHostToDevice);
	cudaCheckErrors("world copy failed");

	free(temp);
}

void BuildWorld(World **h_w, World **d_w, int width,int height){
	
	(*h_w) = (World*)malloc(sizeof(World));
	(*h_w)->vp = (ViewPlane*)malloc(sizeof(ViewPlane));
		
	(*h_w)->vp->hres = width;
	(*h_w)->vp->vres = height;
	(*h_w)->vp->s	 = 1;
	(*h_w)->vp->samplerType = SAMPLER_JITTERED;
	(*h_w)->vp->sampleScale = SAMPLE_SCALE_4;

	(*h_w)->backgroundColor = RGBColor(5,5,30);

	/* GeometricObject
	*/
	(*h_w)->numObject = 5;
	(*h_w)->objects = (GeometricObject **)malloc((*h_w)->numObject * sizeof(GeometricObject*));

	Matte *material1 = newMatte(0.25,0.65,red);	
	Phong *material2 = newPhong(0.25,0.6,green,0.2,20);	
	Phong *material3 = newPhong(0.25,0.6,yellow,0.2,20);	
	Matte *material4 = newMatte(0.25,0.65,white);
	Phong *material5 = newPhong(0.25,0.6,red,0.2,20);

	initSphere( ((Sphere**)((*h_w)->objects)),		Point3D(0,120,280),	120,	(Material*)material5		);
	initSphere( ((Sphere**)((*h_w)->objects+1)),	Point3D(0,150,0),	150,		(Material*)material2		);
	initSphere( ((Sphere**)((*h_w)->objects+2)),	Point3D(210,100,100),	100,		(Material*)material3	);
	initPlane( ((Plane**)((*h_w)->objects+3)),	Point3D(0,0,0),	Normal(0,1,0),		(Material*)material4		);
	initPlane( ((Plane**)((*h_w)->objects+4)),	Point3D(-600,0,-600),	Normal(1,0,1),		(Material*)material4);

/*	(*h_w)->numObject = 100;
	(*h_w)->objects = (GeometricObject **)malloc((*h_w)->numObject * sizeof(GeometricObject*));
	Matte *material = newMatte(0.25,0.65,red);
	Matte *material4 = newMatte(0.25,0.65,RGBColor(5,5,40));

	initPlane(  ((Plane**)((*h_w)->objects )),	Point3D(-600,0,-600),	Normal(1,0,1),		(Material*)material4		);
	for( int i = 1 ; i < (*h_w)->numObject ; ++ i ){
		initSphere( ((Sphere**)((*h_w)->objects + i )), 
			Point3D( 400 * float(rand())/float(RAND_MAX) , 400 * float(rand())/float(RAND_MAX) , 400 * float(rand())/float(RAND_MAX) ),
			35 * float(rand())/float(RAND_MAX) + 5 , (Material*)material); 
	}*/
	
	Pinhole *pinhole = (Pinhole*)malloc(sizeof(Pinhole));
	pinhole->type = CAMARA_TYPE_PINHOLE;
	pinhole->eye = Point3D(500,300,300);
	pinhole->lookat = Point3D(0,100,100);
	pinhole->up = Vector3D(0,1,0);
	pinhole->viewDistance = 400;
	pinhole->zoom = 1;
	ComputeUVW( (Camara*)pinhole );
	(*h_w)->camara = (Camara*)pinhole;

	/* Light
	*/
	Ambient *h_ab = (Ambient*)malloc(sizeof(Ambient));
	h_ab->ls = 0.5;
	h_ab->color = white;
	h_ab->shadows = false;
	h_ab->type = LIGHT_TYPE_AMBIENT;
	(*h_w)->ambient = (Light*)h_ab;

	(*h_w)->numLight = 1;

	(*h_w)->lights = (Light**)malloc( (*h_w)->numLight * sizeof(Light*) );
	
	PointLight *h_pl = (PointLight*)malloc(sizeof(PointLight));
	h_pl->ls = 1;
	h_pl->pos = Point3D(1000,400,1200);
	h_pl->color = white;
	h_pl->shadows = true;
	h_pl->type = LIGHT_TYPE_POINTLIGHT;
	(*h_w)->lights[0] = (Light*)h_pl;
	
	UpdateWorldToDevice(*h_w,d_w);
}

void FreeWorld(World *d_w,World *h_w){


}


__device__ 
void  HitBareBonesObject(World *w, Ray ray,ShadeRec *sr){
	float t;
	float tmin = hugeValue;
		
	sr->hitAnObject = false;

	for(int i = 0 ; i < w->numObject ; ++i){
		if( Hit(w->objects[i],ray,&t,sr) && (t < tmin) ){
			sr->hitAnObject= true;
			tmin = t;
			sr->color = w->objects[i]->color;
		}
	}
}

__device__
void HitObject(World *w, Ray ray, ShadeRec *sr ){
	float t ;
	Normal normal;
	Point3D localHitPoint;
	float tmin = hugeValue;

	for(int i = 0 ; i < w->numObject ; ++i){
		if( Hit( w->objects[i],ray,&t,sr) && (t < tmin )){
			sr->hitAnObject = true;
			tmin = t;
			sr->material = w->objects[i]->material;
			sr->hitPoint = ray.o + t * ray.d;
			normal = sr->normal;
			localHitPoint = sr->localHitPoint;
		}
	}

	if( sr->hitAnObject ){
		//sr->t = tmin;    //ShadeRec does not contain a 't'
		sr->normal = normal;
		sr->localHitPoint = localHitPoint;
	}
}