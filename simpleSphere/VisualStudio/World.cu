#include "defines.cuh"

#include "stdlib.h"
#include "stdio.h"

void build_world(World **w,int width,int height){
	
	World *h_world;
	h_world = (World*)malloc(sizeof(World));
		
	ViewPlane *h_vp = (ViewPlane*)malloc(sizeof(ViewPlane));
	h_vp->hres = width;
	h_vp->vres = height;
	h_vp->s = 1;	
	h_vp->samplerType = SAMPLER_JITTERED;
	h_vp->sampleScale = SAMPLE_SCALE_4;
	hipMalloc((void**)&(h_world->vp),sizeof(ViewPlane));
	cudaCheckErrors("viewplane allocate failed");
	hipMemcpy(h_world->vp,h_vp,sizeof(ViewPlane),hipMemcpyHostToDevice);
	cudaCheckErrors("viewplane memory copy failed");

	h_world->backgroundColor = black;
	
	/********  Geometric Object  ***********/
	h_world->numObject = 4;
	GeometricObject **obj = (GeometricObject **)malloc(h_world->numObject * sizeof(GeometricObject*));

	initSphere( ((Sphere**)(obj)),		Point3D(0,120,280),	120,		red);
	initSphere( ((Sphere**)(obj+1)),	Point3D(0,150,0),	150,		green);
	initSphere( ((Sphere**)(obj+2)),	Point3D(210,100,100),	100,		yellow);
	initPlane( ((Plane**)(obj+3)),	Point3D(0,0,0),	Normal(0,1,0),		blue);

	hipMalloc((void**)&(h_world->object),h_world->numObject  * sizeof(GeometricObject*) );
	cudaCheckErrors("object pointer memory allocate failed");
	
	hipMemcpy(h_world->object,obj,h_world->numObject * sizeof(GeometricObject*),hipMemcpyHostToDevice);
	cudaCheckErrors("object pointer memory copy failed");

	/* camara */
	Pinhole pinhole;
	pinhole.type = CAMARA_TYPE_PINHOLE;
	pinhole.eye = Point3D(300,400,500);
	pinhole.lookat = Point3D(0,100,0);
	pinhole.up = Vector3D(0,1,0);
	pinhole.viewDistance = 50;
	pinhole.zoom = 5;
	ComputeUVW((Camara*)&pinhole);
	hipMalloc((void**)&(h_world->camara),sizeof(Pinhole));
	cudaCheckErrors("pinhole memory allocate failed");
	hipMemcpy((h_world->camara),&pinhole,sizeof(Pinhole),hipMemcpyHostToDevice);
	cudaCheckErrors("pinhole copy failed");

	/*  world */
	hipMalloc((void**) w , sizeof(World));
	cudaCheckErrors( "world allocate failed" );
	hipMemcpy( *w , h_world,sizeof(World),hipMemcpyHostToDevice);
	cudaCheckErrors("world memory copy failed");

	free(h_world);free(h_vp);
}

__device__ 
void  hitBareBonesObject(World *w, Ray ray,ShadeRec *sr){
	float t;
	float tmin = hugeValue;
		
	sr->hitAnObject = false;

	for(int i = 0 ; i < w->numObject ; ++i){
		switch( ((GeometricObject*)(w->object[i]))->type ){
		case GMO_TYPE_SPHERE:
			if( sphereHit((Sphere*)w->object[i],ray,&t,sr) && (t < tmin) ){
				sr->hitAnObject= true;
				tmin = t;
				sr->color = w->object[i]->color;
			}
			break;
		case GMO_TYPE_PLANE:
			if( planeHit((Plane*)w->object[i],ray,&t,sr) && (t < tmin) ){
				sr->hitAnObject= true;
				tmin = t;
				sr->color = w->object[i]->color;
			}
			break;
		default:
			sr->color = w->backgroundColor;
			break;
		}
	}
}