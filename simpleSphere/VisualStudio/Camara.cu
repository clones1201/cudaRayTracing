#include "hip/hip_runtime.h"
#include "defines.cuh"
#include "function_defines.cuh"

__device__ inline
Vector3D RayDirection(Pinhole *ph,Point2D p){
	Vector3D result = p.x * ph->u + p.y * ph->v - ph->w * ph->viewDistance;
	result = Normalize(result);
	return result;
}

__global__ 
void PinholeRenderScene_k(World *w, uchar3 *buffer){
	
	Ray ray;
	ViewPlane vp = *(w->vp);
	Pinhole pinhole = *((Pinhole*)w->camara);
	//int depth = 0;
	
	Point2D sp;		//sample point in [0,1] x [0,1]
	Point2D pp;		//sample point in a pixel

	vp.s = vp.s / pinhole.zoom;
	ray.o = pinhole.eye;

	int r = blockIdx.x * blockDim.x + threadIdx.x;
	int c = blockIdx.y * blockDim.y + threadIdx.y;
	
	int offset = r * gridDim.x * blockDim.x + c;

	int numSample = getSampleNum(  vp.sampleScale  );

	RGBColor pixelColor = black;
	buffer[offset] = make_uchar3(0,0,0);
	for(int i = 0 ; i < numSample ; ++ i ){
			sp = getSampleUnitSquare( vp.samplerType , i , vp.sampleScale );

			//pp.x = w->vp->s * ( c - 0.5 * w->vp->hres + sp.x );
			//pp.y = w->vp->s * ( r - 0.5 * w->vp->vres + sp.y );
			pp.x = vp.s * ( c - 0.5 * vp.hres + sp.x );
			pp.y = vp.s * ( r - 0.5 * vp.vres + sp.y );

			ray.d = RayDirection(&pinhole,pp);

			pixelColor = pixelColor + RayCastTraceRay(w,ray,0) / numSample;
			//singleSphereTraceRay(w,(Sphere*)*(w->object),ray,&pixelColor);
			//pixelColor =multiObjTraceRay(w,ray);
		}
	buffer[offset] = MapToUchar(pixelColor);

}
/*
__global__
	void render_scene_k(World *w,uchar3 *buffer){

		RGBColor pixelColor = red;
		
		Ray ray;
				
		int r = blockIdx.x * blockDim.x + threadIdx.x;
		int c = blockIdx.y * blockDim.y + threadIdx.y;
		
		int offset = r * gridDim.x * blockDim.x + c;

		buffer[offset] = w->backgroundColor;

		Point2D sp;		//sample point in [0,1] x [0,1]
		Point2D pp;		//sample point in a pixel

		//SamplerType type = SAMPLER_JITTERED;		
		//SampleScale scale = SAMPLE_SCALE_16;
		

		int numSample = getSampleNum(  w->vp->sampleScale  );

		for(int i = 0 ; i < numSample ; ++ i ){
			sp = getSampleUnitSquare( w->vp->samplerType , i , w->vp->sampleScale );

			//pp.x = w->vp->s * ( c - 0.5 * w->vp->hres + sp.x );
			//pp.y = w->vp->s * ( r - 0.5 * w->vp->vres + sp.y );
			pp.x = 1 * ( c - 0.5 * 512 + sp.x );
			pp.y = 1 * ( r - 0.5 * 512 + sp.y );

			ray.o = Point3D(pp.x,pp.y,0);
			ray.d = Vector3D(0,0,1);

			//Sphere*s;
			pixelColor = multiObjTraceRay(w,ray);
			//singleSphereTraceRay(w,(Sphere*)*(w->object),ray,&pixelColor);

			buffer[offset] = buffer[offset] + pixelColor / numSample;
			//buffer[offset] = w->backgroundColor;
		}
}
*/

void RenderScene(World *w,int width,int height,uchar3 *buffer){
	
	dim3 blockPerGrid(width/16,height/16);
	dim3 threadPerBlock(16,16);

	World h_world;
	hipMemcpy(&h_world, w,sizeof(World),hipMemcpyDeviceToHost);
	cudaCheckErrors("world copy failed");

	Camara h_camara,*pointer;
	hipMemcpy( &h_camara,h_world.camara,sizeof(Camara),hipMemcpyDeviceToHost);
	cudaCheckErrors("camara copy failed");

	switch( h_camara.type ){
	case CAMARA_TYPE_PINHOLE:
		PinholeRenderScene_k<<<blockPerGrid,threadPerBlock>>>(w,buffer);
		break;
	default:
		break;
	}
/* old version  *//* 
	render_scene_k<<<blockPerGrid,threadPerBlock>>>(w,buffer);
	cudaCheckErrors("render_scene_k failed...");
	*/
}
